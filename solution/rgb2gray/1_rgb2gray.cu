/*
* In this second version of the rgb2gray program, double constants (0.299, 0.587, 0.114) 
* have been replaced with float one (0.299f, 0.587f, 0.114f). The effect is a considerable 
* reduction of the execution time
*/

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include <sys/time.h>

#define CHANNELS 3
#define OUT_FN_CPU "output_cpu.pgm"
#define OUT_FN_GPU "output_gpu.pgm"

inline double milliseconds();
int save_ppm_image(const char* filename, unsigned char* image, unsigned int width, unsigned int height);
int save_pgm_image(const char* filename, unsigned char* image, unsigned int width, unsigned int height);
int load_ppm_image(const char* filename, unsigned char** image, unsigned int* width, unsigned int* height);
int load_pgm_image(const char* filename, unsigned char** image, unsigned int* width, unsigned int* height);


//rgb2gray conversion accelerated onto GPU
__global__ void rgb2grayGPU(unsigned char* input, unsigned char* output, unsigned int width, unsigned int height){
  unsigned char redValue, greenValue, blueValue, grayValue;
  int j = blockIdx.x * blockDim.x + threadIdx.x;
  int i = blockIdx.y * blockDim.y + threadIdx.y;
  
  if(j<width && i<height){
    redValue = input[(i*width + j)*3];
    greenValue = input[(i*width + j)*3+1];
    blueValue = input[(i*width + j)*3+2];
    grayValue = (unsigned char) (0.299f*redValue + 0.587f*greenValue + 0.114f*blueValue);
    output[i*width + j] = grayValue;
  }  
}

int main(int argc, char* argv[]) {
  char* inputfile;
  unsigned int height, width;
  unsigned char *input, *output;
  unsigned char *input_d, *output_d;
  int nPixels;
  int err;

  hipEvent_t gpu_start;
  hipEvent_t gpu_end;
  float gpu_exectime;

  //read arguments
  if(argc!=4){
    printf("Please specify ppm input file name and 2 integer values for X and Y block sizes\n");
    return 0;
  }
  inputfile = argv[1];
  int blockdim_x =atoi(argv[2]);
  int blockdim_y =atoi(argv[3]);

  //load input image
  err = load_ppm_image(inputfile, &input, &width, &height);
  if(err)
    return 1;
  nPixels = width * height;

  //allocate memory for output image
  output = (unsigned char*) malloc(sizeof(unsigned char) * nPixels);
  if(!output){
  	printf("Error with malloc\n");
  	free(input);
  	return 1;
  }
  
  //allocate memory on the GPU
  hipMalloc(&input_d, sizeof(unsigned char) * nPixels*CHANNELS);
  hipMalloc(&output_d, sizeof(unsigned char) * nPixels);
  hipEventCreate(&gpu_start);
  hipEventCreate(&gpu_end);

  //transmit data to GPU
  hipMemcpy(input_d, input, sizeof(unsigned char)*nPixels*CHANNELS, hipMemcpyHostToDevice);

  //invoke the kernel on the GPU
  dim3 blocksPerGrid((width+blockdim_x-1)/blockdim_x, (height+blockdim_y-1)/blockdim_y, 1);
  dim3 threadsPerBlock(blockdim_x, blockdim_y, 1);

  int dev;  
  hipDeviceProp_t deviceProp;
  hipGetDevice(&dev);
  hipGetDeviceProperties(&deviceProp, dev);

  if(threadsPerBlock.x<=0 || threadsPerBlock.x>deviceProp.maxThreadsDim[0] ||
     threadsPerBlock.y<=0 || threadsPerBlock.y>deviceProp.maxThreadsDim[1]) {
    printf("Violated maximum sizes of a dimension of a block (0;%d] - (0:%d] - Specified values: %d %d\n",
                 deviceProp.maxThreadsDim[0], deviceProp.maxThreadsDim[1], 
                 threadsPerBlock.x, threadsPerBlock.y);
    free(input);
    free(output);
    hipDeviceReset();
    return 1;    
  }
  if(threadsPerBlock.x * threadsPerBlock.y > deviceProp.maxThreadsPerBlock){
    printf("Violated maximum number of threads per block (%d) - Specified value: %d\n", 
            deviceProp.maxThreadsPerBlock, threadsPerBlock.x * threadsPerBlock.y);
    free(input);
    free(output);
    hipDeviceReset();
    return 1;    
  }
  
  hipEventRecord(gpu_start);
  rgb2grayGPU<<<blocksPerGrid, threadsPerBlock>>>(input_d, output_d, width, height);
  hipEventRecord(gpu_end);
  hipEventSynchronize(gpu_end);
  //cudaDeviceSynchronize();
  hipEventElapsedTime(&gpu_exectime, gpu_start, gpu_end);

  //transmit data from the GPU
  hipMemcpy(output, output_d, nPixels*sizeof(unsigned char), hipMemcpyDeviceToHost);

  //save output image
  err = save_pgm_image(OUT_FN_GPU, output, width, height);
  if(err){
    free(input);
    free(output);
    hipDeviceReset();
    return 1;
  }
  
  printf("Execution time: %f\n", gpu_exectime);

  //cleanup
  free(input);
  free(output);
  hipFree(input_d);
  hipFree(output_d);
  hipEventDestroy(gpu_start);
  hipEventDestroy(gpu_end);

  return 0;
}

int save_ppm_image(const char* filename, unsigned char* image, unsigned int width, unsigned int height) {
  FILE *f; //output file handle

  //open the output file and write header info for PPM filetype
  f = fopen(filename, "wb");
  if (f == NULL){
    fprintf(stderr, "Error opening 'output.ppm' output file\n");
    return -1;
  }
  fprintf(f, "P6\n");
  fprintf(f, "%d %d\n%d\n", width, height, 255);
  fwrite(image, sizeof(unsigned char), height*width*CHANNELS, f);
  fclose(f);
  return 0;
}

int save_pgm_image(const char* filename, unsigned char* image, unsigned int width, unsigned int height) {
  FILE *f; //output file handle

  //open the output file and write header info for PPM filetype
  f = fopen(filename, "wb");
  if (f == NULL){
    fprintf(stderr, "Error opening 'output.ppm' output file\n");
    return -1;
  }
  fprintf(f, "P5\n");
  fprintf(f, "%d %d\n%d\n", width, height, 255);
  fwrite(image, sizeof(unsigned char), height*width, f);
  fclose(f);
  return 0;
}


int load_ppm_image(const char* filename, unsigned char** image, unsigned int* width, unsigned int* height) {
  FILE *f; //input file handle
  char temp[256];
  unsigned int s;

  //open the input file and write header info for PPM filetype
  f = fopen(filename, "rb");
  if (f == NULL){
    fprintf(stderr, "Error opening '%s' input file\n", filename);
    return -1;
  }
  fscanf(f, "%s\n", temp);
  fscanf(f, "%d %d\n", width, height);
  fscanf(f, "%d\n",&s);

  *image = (unsigned char*) malloc(sizeof(unsigned char)* (*width) * (*height) * CHANNELS);
  if(*image)
	  fread(*image, sizeof(unsigned char), (*width) * (*height) * CHANNELS, f);
  else{
  	printf("Error with malloc\n");
  	return -1;
  }

  fclose(f);
  return 0;
}

int load_pgm_image(const char* filename, unsigned char** image, unsigned int* width, unsigned int* height) {
  FILE *f; //input file handle
  char temp[256];
  unsigned int s;

  //open the input file and write header info for PPM filetype
  f = fopen(filename, "rb");
  if (f == NULL){
    fprintf(stderr, "Error opening '%s' input file\n", filename);
    return -1;
  }
  fscanf(f, "%s\n", temp);
  fscanf(f, "%d %d\n", width, height);
  fscanf(f, "%d\n",&s);

  *image = (unsigned char*) malloc(sizeof(unsigned char)* (*width) * (*height));
  if(*image)
    fread(*image, sizeof(unsigned char), (*width) * (*height), f);
  else{
    printf("Error with malloc\n");
    return -1;
  }

  fclose(f);
  return 0;
}

inline double milliseconds(){
  struct timeval tp;
  struct timezone tzp;
  int i = gettimeofday(&tp, &tzp);
  return ((double)tp.tv_sec * 1000 + (double)tp.tv_usec * 0.001);
}

