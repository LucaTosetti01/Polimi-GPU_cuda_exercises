
#include <hip/hip_runtime.h>
#include <chrono>
#include <cmath>
#include <iostream>
#include <vector>

#define DIM 512
#define BLOCK_DIM 256

#define STRIDE_FACTOR 2
// CPU version of the reduction kernel
float reduce_cpu(const double *data, const int length)
{
    float sum = 0;
    for (int i = 0; i < length; i++)
    {
        sum += data[i];
    }
    return sum;
}

__global__ void reduce_gpu(double *__restrict__ input, double *__restrict__ output)
{
    const unsigned int i = threadIdx.x;

    // Apply the offset
    input += blockDim.x * blockIdx.x * STRIDE_FACTOR;
    output += blockIdx.x;

    __shared__ double shared_res[BLOCK_DIM];

    for (unsigned int iter = 0; iter < DIM; iter += blockDim.x)
    {
        shared_res[i] += input[i + iter];
    }
   //shared_res[i] = input[i] + input[i + BLOCK_DIM];
    __syncthreads();
    for (unsigned int stride = blockDim.x / STRIDE_FACTOR; stride >= 1; stride /= STRIDE_FACTOR)
    {
        if (i < stride)
        {
            shared_res[i] += shared_res[i + stride];
        }

        __syncthreads();
    }

    __syncthreads();
    // Write result for this block to global memory
    if (i == 0)
    {
        // You could have used only a single memory location and performed an atomicAdd
        *output = shared_res[0];
    }
}

int main()
{
    std::vector<double> data(DIM);
    for (int i = 0; i < DIM; ++i)
        data[i] = static_cast<double>(rand()) / RAND_MAX; // Random value between 0 and 1 }

    // CPU version
    double sum_cpu = reduce_cpu(data.data(), data.size());

    std::cout << "Reduction CPU is " << sum_cpu << std::endl;

    double *d_data;
    double *d_output;

    hipMalloc(&d_data, sizeof(double) * DIM);
    hipMemcpy(d_data, data.data(), sizeof(double) * DIM, hipMemcpyHostToDevice);

    dim3 threadsPerBlock(BLOCK_DIM, 1, 1);
    dim3 blocksPerGrid((DIM - 1) / (BLOCK_DIM * STRIDE_FACTOR) + 1, 1, 1);

    hipMalloc(&d_output, sizeof(double) * blocksPerGrid.x);
    reduce_gpu<<<blocksPerGrid, threadsPerBlock>>>(d_data, d_output);
    hipDeviceSynchronize();

    double *gpu_results = (double *)malloc(sizeof(double) * blocksPerGrid.x);
    hipMemcpy(gpu_results, d_output, sizeof(double) * blocksPerGrid.x, hipMemcpyDeviceToHost);
    double sum_gpu = 0;

    for (int i = 0; i < blocksPerGrid.x; i++)
    {
        sum_gpu += gpu_results[i];
    }

    std::cout << "Reduction GPU is " << sum_gpu << std::endl;

    if (std::abs(sum_cpu - sum_gpu) > 1e-3)
    {
        std::cout << "Reduction CPU and GPU are NOT equivalent!" << std::endl;
        std::cout << "CPU: " << sum_cpu << std::endl;
        std::cout << "GPU: " << sum_gpu << std::endl;
        return EXIT_FAILURE;
    }

    std::cout << "Reduction CPU and GPU are equivalent!" << std::endl;

    // Cleanup
    hipFree(d_data);
    hipFree(d_output);
    return 0;
}
