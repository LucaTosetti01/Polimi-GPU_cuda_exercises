#include "hip/hip_runtime.h"
#include <iostream>
#include <cmath>
#include <cstdlib>
#include <cassert>

using input_type = float;
using filter_type = input_type;

#define FILTER_RADIUS 4
#define FILTER_SIZE (FILTER_RADIUS * 2 + 1)

__constant__ filter_type filter_const[FILTER_SIZE * FILTER_SIZE];

#define TILE_DIM 32

void convolution_cpu(input_type *input, const input_type *filter, input_type *output, const int width, const int height, const int filter_size, const int filter_radius)
{
    for (int outRow = 0; outRow < width; outRow++)
    {
        for (int outCol = 0; outCol < height; outCol++)
        {
            input_type value{0.0f};
            for (int row = 0; row < filter_size; row++)
                for (int col = 0; col < filter_size; col++)
                {
                    int inRow = outRow - filter_radius + row;
                    int inCol = outCol - filter_radius + col;
                    if (inRow >= 0 && inRow < height && inCol >= 0 && inCol < width)
                    {
                        value += filter[row * filter_size + col] * input[inRow * width + inCol];
                    }
                }
            output[outRow * width + outCol] = value;
        }
    }
}

__global__ void convolution_gpu(input_type *input, input_type *output, const int width, const int height, const int filter_size, const int filter_radius)
{
    const int col = blockIdx.x * TILE_DIM + threadIdx.x;
    const int row = blockIdx.y * TILE_DIM + threadIdx.y;

    __shared__ input_type tile_s[TILE_DIM][TILE_DIM];
    if (row < height && col < width)
    {
        tile_s[threadIdx.y][threadIdx.x] = input[row * width + col];
    }
    else
    {
        tile_s[threadIdx.y][threadIdx.x] = 0.0;
    }

    __syncthreads();

    if (col < width && row < height)
    {
        float PValue = 0.0f;
        for (int fRow = 0; fRow < FILTER_SIZE; fRow++)
        {
            for (int fCol = 0; fCol < FILTER_SIZE; fCol++)
            {
                int currTileRow = threadIdx.y - FILTER_RADIUS + fRow;
                int currTileCol = threadIdx.x - FILTER_RADIUS + fCol;
                if (currTileCol >= 0 && currTileCol < TILE_DIM &&
                    currTileRow >= 0 && currTileRow < TILE_DIM)
                    PValue += filter_const[fRow * FILTER_SIZE + fCol] * tile_s[currTileRow][currTileCol];
                else
                {
                    if (row - FILTER_RADIUS + fRow >= 0 && row - FILTER_RADIUS + fRow < width &&
                        col - FILTER_RADIUS + fCol >= 0 && col - FILTER_RADIUS + fCol < height)
                    {
                        PValue += filter_const[fRow * FILTER_SIZE + fCol] * input[row * width + col];
                    }
                }
            }
        }
        output[row * width + col] = PValue;
    }
}

int main(int argc, char **argv)
{
    if (argc != 2)
    {
        printf("Please specify matrix dimensions\n");
        return EXIT_FAILURE;
    }
    const unsigned dim = atoi(argv[1]);
    const unsigned int width = dim;
    const unsigned int height = dim;

    input_type *input = new input_type[width * height];               // Input
    filter_type *filter = new filter_type[FILTER_SIZE * FILTER_SIZE]; // Convolution filter
    input_type *output_cpu = new input_type[width * height];          // Output (CPU)
    input_type *output_gpu = new input_type[width * height];          // Output (GPU)

    // Randomly initialize the inputs
    for (int i = 0; i < FILTER_SIZE * FILTER_SIZE; i++)
        filter[i] = static_cast<filter_type>(rand()) / RAND_MAX;

    for (int i = 0; i < width * height; ++i)
        input[i] = static_cast<input_type>(rand()) / RAND_MAX; // Random value between 0 and 1

    // Call CPU convolution
    convolution_cpu(input, filter, output_cpu, width, height, FILTER_SIZE, FILTER_RADIUS);

    input_type *d_input;
    filter_type *d_filter;
    input_type *d_output;
    hipMalloc(&d_input, sizeof(input_type) * width * height);
    hipMalloc(&d_filter, sizeof(filter_type) * FILTER_SIZE * FILTER_SIZE);
    hipMalloc(&d_output, sizeof(input_type) * width * height);

    hipMemcpy(d_input, input, sizeof(input_type) * width * height, hipMemcpyHostToDevice);
    hipMemcpy(d_filter, filter, sizeof(filter_type) * FILTER_SIZE * FILTER_SIZE, hipMemcpyHostToDevice);

    dim3 threadsPerBlock(IN_TILE_DIM, IN_TILE_DIM, 1);
    dim3 blocksPerGrid(width / (threadsPerBlock.x - 2 * FILTER_RADIUS) + 1,
                       height / (threadsPerBlock.y - 2 * FILTER_RADIUS) + 1, 1);

    hipMemcpyToSymbol(HIP_SYMBOL(filter_const), filter, FILTER_SIZE * FILTER_SIZE * sizeof(filter_type));
    convolution_gpu<<<blocksPerGrid, threadsPerBlock>>>(d_input, d_output, width, height, FILTER_SIZE, FILTER_RADIUS);
    hipDeviceSynchronize();

    hipMemcpy(output_gpu, d_output, sizeof(input_type) * width * height, hipMemcpyDeviceToHost);

    for (int i = 0; i < height; i++)
    {
        for (int j = 0; j < width; j++)
        {
            if (std::abs(output_cpu[i * width + j] - output_gpu[i * width + j]) > 1e-3)
            {
                printf("Results NOT correct!\n");
                return 1;
            }
        }
    }
    printf("ALL OK...\n");
    // Cleanup and deallocate memory
    delete[] input;
    delete[] filter;
    delete[] output_cpu;
    delete[] output_gpu;

    hipDeviceReset();
    return EXIT_SUCCESS;
}