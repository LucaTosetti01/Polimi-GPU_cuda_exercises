#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include <sys/time.h>

#define CHANNELS 3
#define OUT_FN_CPU "output_cpu.pgm"
#define OUT_FN_GPU "output_gpu.pgm"
#define BLURDIM 10

inline double milliseconds();
int save_ppm_image(const char *filename, unsigned char *image, unsigned int width, unsigned int height);
int save_pgm_image(const char *filename, unsigned char *image, unsigned int width, unsigned int height);
int load_ppm_image(const char *filename, unsigned char **image, unsigned int *width, unsigned int *height);
int load_pgm_image(const char *filename, unsigned char **image, unsigned int *width, unsigned int *height);
void rgb2gray(unsigned char *input, unsigned char *output, unsigned int width, unsigned int height);
void blur(unsigned char *input, unsigned char *output, unsigned int width, unsigned int height);

void rgb2gray(unsigned char *input, unsigned char *output, unsigned int width, unsigned int height)
{
  int i, j;
  unsigned char redValue, greenValue, blueValue, grayValue;
  // loop on all pixels and convert from RGB to gray scale
  for (i = 0; i < height; i++)
  {
    for (j = 0; j < width; j++)
    {
      redValue = input[(i * width + j) * 3];
      greenValue = input[(i * width + j) * 3 + 1];
      blueValue = input[(i * width + j) * 3 + 2];
      grayValue = (unsigned char)(0.299 * redValue + 0.587 * greenValue + 0.114 * blueValue);
      output[i * width + j] = grayValue;
    }
  }
}

void blur(unsigned char *input, unsigned char *output, unsigned int width, unsigned int height)
{
  int i, j, h, k, sum, count;
  // loop on all pixels and to compute the mean value of the intensity together with the 8 neighbor pixels
  for (i = 0; i < height; i++)
  {
    for (j = 0; j < width; j++)
    {
      count = 0;
      sum = 0;
      for (h = -BLURDIM; h <= BLURDIM; h++)
        for (k = -BLURDIM; k <= BLURDIM; k++)
          if (i + h >= 0 && i + h < height && j + k >= 0 && j + k < width)
          {
            count++;
            sum = sum + input[(i + h) * width + (j + k)];
          }
      output[i * width + j] = (float)sum / count;
    }
  }
}

__global__ void rgb2grayGPU(unsigned char *input, unsigned char *output, unsigned int width, unsigned int height)
{
  unsigned char redValue, greenValue, blueValue, grayValue;
  int j = blockIdx.x * blockDim.x + threadIdx.x;
  int i = blockIdx.y * blockDim.y + threadIdx.y;

  if (j < width && i < height)
  {
    redValue = input[(i * width + j) * 3];
    greenValue = input[(i * width + j) * 3 + 1];
    blueValue = input[(i * width + j) * 3 + 2];
    grayValue = (unsigned char)(0.299 * redValue + 0.587 * greenValue + 0.114 * blueValue);
    output[i * width + j] = grayValue;
  }
}

__global__ void blurGPU(unsigned char *input, unsigned char *output, unsigned int width, unsigned int height)
{
  int sum, count, h, k;
  int j = blockIdx.x * blockDim.x + threadIdx.x;
  int i = blockIdx.y * blockDim.y + threadIdx.y;

  count = 0;
  sum = 0;
  if (j < width && i < height)
  {
    for (h = -BLURDIM; h <= BLURDIM; h++)
      for (k = -BLURDIM; k <= BLURDIM; k++)
        if (i + h >= 0 && i + h < height && j + k >= 0 && j + k < width)
        {
          count++;
          sum = sum + input[(i + h) * width + (j + k)];
        }
    output[i * width + j] = (float)sum / count;
  }
}

int main(int argc, char *argv[])
{
  char *inputfile;
  unsigned int height, width;
  unsigned char *input, *gray, *output;
  unsigned char *input_d, *gray_d, *output_d;
  int nPixels;
  int err;

  double cpu_start, cpu_end, cpu_exectime;
  hipEvent_t gpu_start;
  hipEvent_t gpu_end;
  float gpu_exectime;

  // read arguments
  if (argc != 4)
  {
    printf("Please specify ppm input file name and 2 integer values for X and Y block sizes\n");
    return 0;
  }
  inputfile = argv[1];
  int blockdim_x = atoi(argv[2]);
  int blockdim_y = atoi(argv[3]);

  // load input image
  err = load_ppm_image(inputfile, &input, &width, &height);
  if (err)
    return 1;
  nPixels = width * height;

  // allocate memory for gray image
  gray = (unsigned char *)malloc(sizeof(unsigned char) * nPixels);
  if (!gray)
  {
    printf("Error with malloc\n");
    free(input);
    return 1;
  }

  // allocate memory for output image
  output = (unsigned char *)malloc(sizeof(unsigned char) * nPixels);
  if (!output)
  {
    printf("Error with malloc\n");
    free(input);
    return 1;
  }

  // process image on CPU
  cpu_start = milliseconds();
  rgb2gray(input, gray, width, height);
  blur(gray, output, width, height);
  cpu_end = milliseconds();
  cpu_exectime = cpu_end - cpu_start;

  // save output image
  err = save_pgm_image(OUT_FN_CPU, output, width, height);
  if (err)
  {
    free(input);
    free(output);
    return 1;
  }

  // allocate memory on the GPU
  hipMalloc(&input_d, sizeof(unsigned char) * nPixels * CHANNELS);
  hipMalloc(&gray_d, sizeof(unsigned char) * nPixels);
  hipMalloc(&output_d, sizeof(unsigned char) * nPixels);
  hipEventCreate(&gpu_start);
  hipEventCreate(&gpu_end);

  // transmit data to GPU
  hipMemcpy(input_d, input, sizeof(unsigned char) * nPixels * CHANNELS, hipMemcpyHostToDevice);

  // invoke the kernel on the GPU
  dim3 blocksPerGrid((width + blockdim_x - 1) / blockdim_x, (height + blockdim_y - 1) / blockdim_y, 1);
  dim3 threadsPerBlock(blockdim_x, blockdim_y, 1);

  int dev;
  hipDeviceProp_t deviceProp;
  hipGetDevice(&dev);
  hipGetDeviceProperties(&deviceProp, dev);

  if (threadsPerBlock.x <= 0 || threadsPerBlock.x > deviceProp.maxThreadsDim[0] ||
      threadsPerBlock.y <= 0 || threadsPerBlock.y > deviceProp.maxThreadsDim[1])
  {
    printf("Violated maximum sizes of a dimension of a block (0;%d] - (0:%d] - Specified values: %d %d\n",
           deviceProp.maxThreadsDim[0], deviceProp.maxThreadsDim[1],
           threadsPerBlock.x, threadsPerBlock.y);
    free(input);
    free(output);
    hipDeviceReset();
    return 1;
  }
  if (threadsPerBlock.x * threadsPerBlock.y > deviceProp.maxThreadsPerBlock)
  {
    printf("Violated maximum number of threads per block (%d) - Specified value: %d\n",
           deviceProp.maxThreadsPerBlock, threadsPerBlock.x * threadsPerBlock.y);
    free(input);
    free(output);
    hipDeviceReset();
    return 1;
  }

  hipEventRecord(gpu_start);
  rgb2grayGPU<<<blocksPerGrid, threadsPerBlock>>>(input_d, gray_d, width, height);
  blurGPU<<<blocksPerGrid, threadsPerBlock>>>(gray_d, output_d, width, height);
  hipEventRecord(gpu_end);
  hipEventSynchronize(gpu_end);
  // cudaDeviceSynchronize();
  hipEventElapsedTime(&gpu_exectime, gpu_start, gpu_end);

  // transmit data from the GPU
  hipMemcpy(output, output_d, nPixels * sizeof(unsigned char), hipMemcpyDeviceToHost);

  // save output image
  err = save_pgm_image(OUT_FN_GPU, output, width, height);
  if (err)
  {
    free(input);
    free(output);
    hipDeviceReset();
    return 1;
  }

  printf("Execution times\n- CPU: %f\n- GPU: %f\n", cpu_exectime, gpu_exectime);

  // cleanup
  free(input);
  free(gray);
  free(output);
  hipFree(input_d);
  hipFree(gray_d);
  hipFree(output_d);
  hipEventDestroy(gpu_start);
  hipEventDestroy(gpu_end);

  return 0;
}

int save_ppm_image(const char *filename, unsigned char *image, unsigned int width, unsigned int height)
{
  FILE *f; // output file handle

  // open the output file and write header info for PPM filetype
  f = fopen(filename, "wb");
  if (f == NULL)
  {
    fprintf(stderr, "Error opening 'output.ppm' output file\n");
    return -1;
  }
  fprintf(f, "P6\n");
  fprintf(f, "%d %d\n%d\n", width, height, 255);
  fwrite(image, sizeof(unsigned char), height * width * CHANNELS, f);
  fclose(f);
  return 0;
}

int save_pgm_image(const char *filename, unsigned char *image, unsigned int width, unsigned int height)
{
  FILE *f; // output file handle

  // open the output file and write header info for PPM filetype
  f = fopen(filename, "wb");
  if (f == NULL)
  {
    fprintf(stderr, "Error opening 'output.ppm' output file\n");
    return -1;
  }
  fprintf(f, "P5\n");
  fprintf(f, "%d %d\n%d\n", width, height, 255);
  fwrite(image, sizeof(unsigned char), height * width, f);
  fclose(f);
  return 0;
}

int load_ppm_image(const char *filename, unsigned char **image, unsigned int *width, unsigned int *height)
{
  FILE *f; // input file handle
  char temp[256];
  unsigned int s;

  // open the input file and write header info for PPM filetype
  f = fopen(filename, "rb");
  if (f == NULL)
  {
    fprintf(stderr, "Error opening '%s' input file\n", filename);
    return -1;
  }
  fscanf(f, "%s\n", temp);
  fscanf(f, "%d %d\n", width, height);
  fscanf(f, "%d\n", &s);

  *image = (unsigned char *)malloc(sizeof(unsigned char) * (*width) * (*height) * CHANNELS);
  if (*image)
    fread(*image, sizeof(unsigned char), (*width) * (*height) * CHANNELS, f);
  else
  {
    printf("Error with malloc\n");
    return -1;
  }

  fclose(f);
  return 0;
}

int load_pgm_image(const char *filename, unsigned char **image, unsigned int *width, unsigned int *height)
{
  FILE *f; // input file handle
  char temp[256];
  unsigned int s;

  // open the input file and write header info for PPM filetype
  f = fopen(filename, "rb");
  if (f == NULL)
  {
    fprintf(stderr, "Error opening '%s' input file\n", filename);
    return -1;
  }
  fscanf(f, "%s\n", temp);
  fscanf(f, "%d %d\n", width, height);
  fscanf(f, "%d\n", &s);

  *image = (unsigned char *)malloc(sizeof(unsigned char) * (*width) * (*height));
  if (*image)
    fread(*image, sizeof(unsigned char), (*width) * (*height), f);
  else
  {
    printf("Error with malloc\n");
    return -1;
  }

  fclose(f);
  return 0;
}

inline double milliseconds()
{
  struct timeval tp;
  struct timezone tzp;
  int i = gettimeofday(&tp, &tzp);
  return ((double)tp.tv_sec * 1000 + (double)tp.tv_usec * 0.001);
}
