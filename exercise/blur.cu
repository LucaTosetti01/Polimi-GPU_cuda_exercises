#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>

#define CHANNELS 3
#define OUT_FN_CPU "output.pgm"
#define BLURDIM 10

/*Returns the current system time in milliseconds (declared as an inline function, so that the
  GCC compiler is able to make calls to that function faster, see: https://gcc.gnu.org/onlinedocs/gcc/Inline.html)*/
inline double milliseconds()
{
    struct timeval tp;
    struct timezone tzp;
    int i = gettimeofday(&tp, &tzp);
    return ((double)tp.tv_sec * 1000 + (double)tp.tv_usec * 0.001);
}

/*ENUM representing the different image types usable (added for steamline and reuse the code)*/
enum class IMAGE_TYPE
{
    ppm,
    pgm
};

int save_image(const char *filename, unsigned char *image, unsigned int width, unsigned int height, IMAGE_TYPE img_type);
int load_image(const char *filename, unsigned char **image, unsigned int *width, unsigned int *height, IMAGE_TYPE img_type);
void rgb2gray(unsigned char *input, unsigned char *output, unsigned int width, unsigned int height);
void blur(unsigned char *input, unsigned char *output, unsigned int width, unsigned int height);

__global__ void rgb2grayKernel(unsigned char *input, unsigned char *output, unsigned int width, unsigned int height)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned char redValue, greenValue, blueValue, grayValue;

    if (i < height && j < width)
    {
        redValue = input[(i * width + j) * 3];
        greenValue = input[(i * width + j) * 3 + 1];
        blueValue = input[(i * width + j) * 3 + 2];
        grayValue = (unsigned char)(0.299 * redValue + 0.587 * greenValue + 0.114 * blueValue);
        output[i * width + j] = grayValue;
    }
}

__global__ void blurKernel(unsigned char *input, unsigned char *output, unsigned int width, unsigned int height)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    int h, k, sum = 0, count = 0;
    if (j < width && i < height)
    {
        for (h = -BLURDIM; h <= BLURDIM; h++)
            for (k = -BLURDIM; k <= BLURDIM; k++)
                if (i + h >= 0 && i + h < height && j + k >= 0 && j + k < width)
                {
                    count++;
                    sum = sum + input[(i + h) * width + (j + k)];
                }
        output[i * width + j] = (float)sum / count;
    }
}

void rgb2gray(unsigned char *input, unsigned char *output, unsigned int width, unsigned int height)
{
    int i, j;
    unsigned char redValue, greenValue, blueValue, grayValue;
    // loop on all pixels and convert from RGB to gray scale
    for (i = 0; i < height; i++)
    {
        for (j = 0; j < width; j++)
        {
            redValue = input[(i * width + j) * 3];
            greenValue = input[(i * width + j) * 3 + 1];
            blueValue = input[(i * width + j) * 3 + 2];
            grayValue = (unsigned char)(0.299 * redValue + 0.587 * greenValue + 0.114 * blueValue);
            output[i * width + j] = grayValue;
        }
    }
}

void blur(unsigned char *input, unsigned char *output, unsigned int width, unsigned int height)
{
    int i, j, h, k, sum, count;
    // loop on all pixels and to compute the mean value of the intensity together with the 8 neighbor pixels
    for (i = 0; i < height; i++)
    {
        for (j = 0; j < width; j++)
        {
            count = 0;
            sum = 0;
            for (h = -BLURDIM; h <= BLURDIM; h++)
                for (k = -BLURDIM; k <= BLURDIM; k++)
                    if (i + h >= 0 && i + h < height && j + k >= 0 && j + k < width)
                    {
                        count++;
                        sum = sum + input[(i + h) * width + (j + k)];
                    }
            output[i * width + j] = (float)sum / count;
        }
    }
}

int main(int argc, char *argv[])
{
    char *inputfile;
    unsigned int height, width;
    unsigned char *input, *gray, *output;
    unsigned char *d_input, *d_gray, *d_output;
    int nPixels;
    int err;

    double cpu_start, cpu_end, cpu_exectime;
    hipEvent_t gpu_start;
    hipEvent_t gpu_end;
    float gpu_exectime;

    // read arguments
    if (argc != 4)
    {
        printf("Please specify ppm input file name\n");
        return 0;
    }
    inputfile = argv[1];
    int blockDimX = atoi(argv[2]);
    int blockDimY = atoi(argv[3]);

    // load input image
    err = load_image(inputfile, &input, &width, &height, IMAGE_TYPE::ppm);

    if (err)
        return 1;
    nPixels = width * height;
    // allocate memory for gray image
    gray = (unsigned char *)malloc(sizeof(unsigned char) * nPixels);
    if (!gray)
    {
        printf("Error with malloc\n");
        free(input);
        return 1;
    }

    // allocate memory for output image
    output = (unsigned char *)malloc(sizeof(unsigned char) * nPixels);
    if (!output)
    {
        printf("Error with malloc\n");
        free(gray);
        free(input);
        return 1;
    }

    // process image
    cpu_start = milliseconds();
    rgb2gray(input, gray, width, height);
    blur(gray, output, width, height);
    cpu_end = milliseconds();
    cpu_exectime = cpu_end - cpu_start;

    // save output image
    err = save_image(OUT_FN_CPU, output, width, height, IMAGE_TYPE::pgm);
    if (err)
    {
        free(input);
        free(gray);
        free(output);
        return 1;
    }

    /*Allocating memory on the Device for the input, intermediate result (gray) and  output arrays*/
    hipMalloc(&d_input, sizeof(unsigned char) * nPixels * CHANNELS);
    hipMalloc(&d_gray, sizeof(unsigned char) * nPixels);
    hipMalloc(&d_output, sizeof(unsigned char) * nPixels);
    /*Creating gpu_start and gpu_end events in order to use them for recording kernel execution duration*/
    hipEventCreate(&gpu_start);
    hipEventCreate(&gpu_end);

    /*Copying the input data from the Host to the Device array*/
    hipMemcpy(d_input, input, sizeof(unsigned char) * nPixels * CHANNELS, hipMemcpyHostToDevice);

    /*Declaring the blocks per grid and threads per block strcuts by using the command line parameters read before*/
    dim3 blocksPerGrid(ceil((float)width / blockDimX), ceil((float)height / blockDimY), 1);
    dim3 threadsPerBlock(blockDimX, blockDimY, 1);

    /*Getting the Device's properties in order to check that the inserted grid and block dimensions don't exceed the number of
      threads per block and the number of total threads allowed by the current Device*/
    int dev;
    hipDeviceProp_t deviceProp;
    hipGetDevice(&dev);
    hipGetDeviceProperties(&deviceProp, dev);

    /*Checks whether the block size command line parameters don't violate some device's dimension properties*/
    if (threadsPerBlock.x <= 0 || threadsPerBlock.x > deviceProp.maxThreadsDim[0] ||
        threadsPerBlock.y <= 0 || threadsPerBlock.y > deviceProp.maxThreadsDim[1])
    {
        printf("Violeted the minimum or maximum size of the dimension of a block(0;%d] - (0;%d]",
               deviceProp.maxThreadsDim[0],
               deviceProp.maxThreadsDim[1]);
        free(input);
        free(output);
        free(gray);
        hipDeviceReset();
        return 1;
    }

    if (threadsPerBlock.x * threadsPerBlock.y > deviceProp.maxThreadsPerBlock)
    {
        printf("Violeted the maximum number of threads per block(0;%d]",
               deviceProp.maxThreadsPerBlock);
        free(input);
        free(output);
        free(gray);
        hipDeviceReset();
        return 1;
    }

    /*Record gpu_start event in order to "start the timer" with which we will compute the Device's execution time*/
    hipEventRecord(gpu_start);
    rgb2grayKernel<<<blocksPerGrid, threadsPerBlock>>>(d_input, d_gray, width, height);
    blurKernel<<<blocksPerGrid, threadsPerBlock>>>(d_gray, d_output, width, height);
    /*Record gpu_end event in order to "end the timer" with which we will compute the Device's execution time*/
    hipEventRecord(gpu_end);
    /*Waiting that the kernel execution has ended and that the gpu_end event has been processed by the Device
      (thing that will happen after the end of the rgb2grayKernel and blurKernel functions)*/
    hipEventSynchronize(gpu_end);
    /*Compute execution time of the Device*/
    hipEventElapsedTime(&gpu_exectime, gpu_start, gpu_end);

    /*Copying the output of the blur process from the Device memory to the Host memory*/
    hipMemcpy(output, d_output, sizeof(unsigned char) * nPixels, hipMemcpyDeviceToHost);

    // save output image
    err = save_image(OUT_FN_CPU, output, width, height, IMAGE_TYPE::pgm);
    if (err)
    {
        free(input);
        free(gray);
        free(output);
        hipDeviceReset();
        return 1;
    }

    /*Print some time comparisons*/
    printf("Host's execution time: %f\n", cpu_exectime);
    printf("Kernel function's execution time: %f\n", gpu_exectime);
    bool ifDeviceFaster = gpu_exectime < cpu_exectime ? 1 : 0;
    printf("%s was %f milliseconds faster!\n", (ifDeviceFaster ? "Device" : "Host"), (ifDeviceFaster ? cpu_exectime - gpu_exectime : gpu_exectime - cpu_exectime));

    // cleanup
    free(input);
    free(gray);
    free(output);
    hipDeviceReset();

    return 0;
}

int save_image(const char *filename, unsigned char *image, unsigned int width, unsigned int height, IMAGE_TYPE img_type)
{
    FILE *f; // output file handle

    // open the output file and write header info for PPM filetype
    f = fopen(filename, "wb");
    if (f == NULL)
    {
        fprintf(stderr, "Error opening 'output.ppm' output file\n");
        return -1;
    }
    if (img_type == IMAGE_TYPE::ppm)
        fprintf(f, "P6\n");
    else
        fprintf(f, "P5\n");

    fprintf(f, "%d %d\n%d\n", width, height, 255);
    fwrite(image, sizeof(unsigned char), height * width * ((img_type == IMAGE_TYPE::ppm) ? CHANNELS : 1), f);
    fclose(f);
    return 0;
}

int load_image(const char *filename, unsigned char **image, unsigned int *width, unsigned int *height, IMAGE_TYPE img_type)
{
    FILE *f; // input file handle
    char temp[256];
    unsigned int s;

    // open the input file and write header info for PPM filetype
    f = fopen(filename, "rb");
    if (f == NULL)
    {
        fprintf(stderr, "Error opening '%s' input file\n", filename);
        return -1;
    }
    fscanf(f, "%s\n", temp);
    fscanf(f, "%d %d\n", width, height);
    fscanf(f, "%d\n", &s);

    *image = (unsigned char *)malloc(sizeof(unsigned char) * (*width) * (*height) * ((img_type == IMAGE_TYPE::ppm) ? CHANNELS : 1));
    if (*image)
        fread(*image, sizeof(unsigned char), (*width) * (*height) * CHANNELS, f);
    else
    {
        printf("Error with malloc\n");
        return -1;
    }

    fclose(f);
    return 0;
}